#include "hip/hip_runtime.h"
#ifndef _SIMPLED3D_KERNEL_CU_
#define _SIMPLED3D_KERNEL_CU_

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <builtin_types.h>
#include <hip/hip_runtime_api.h>

#include <cutil_inline.h>

///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param pos  pos in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void kernel( float4* pos, unsigned int width, unsigned int height, float time)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // calculate uv coordinates
    float u = x / (float) width;
    float v = y / (float) height;
    u = u*2.0f - 1.0f;
    v = v*2.0f - 1.0f;

    // calculate simple sine wave pattern
    float freq = 4.0f;
    float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;

    // write output vertex
    pos[y*width+x] = make_float4(u, w, v, __int_as_float(0xff00ff00));
}

extern "C" 
void sort_faces(float* vertex_buffer, float* index_buffer, float3 sorting_argument, uint3 *objects, uint1 count, float* tranformations){
    /*hipError_t error = hipSuccess;

    dim3 block(8, 8, 1);
    dim3 grid(width / block.x, height / block.y, 1);

	kernel<<<grid, block>>>(pos, width, height, time);

    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("kernel() failed to launch error = %d\n", error);
    }*/
}

#endif
